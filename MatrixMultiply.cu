#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//Matrix multiplication kernel - thread specification
__global__
void MatrixMulKernel(float* Md, float* Nd, float* Pd, int width){
  //2D Thread ID
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  //Pvalue stores the Pd element that is computed by the thread
  float Mdelement, Ndelement, Pvalue = 0;
  for(int k = 0; k < width ; ++k){
    Mdelement = Md[ty*width + k];
    Ndelement = Nd[k*width + tx];
    Pvalue += (Mdelement*Ndelement);
  }
  Pd[ty*width + tx] = Pvalue;
}
void MatrixMultiplication(float* M, float* N, float* P, int width){
  int size = width*width*sizeof(float);
  float* Md, *Nd, *Pd;
  //Transfer M and N to device memory
  hipMalloc((void**)&Md, size);
  hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
  hipMalloc((void**)&Nd, size);
  hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
  //Allocate P on the device
  hipMalloc((void**)&Pd, size);
  //Setup the execution configuration
  dim3 dimBlock(width, width);
  dim3 dimGrid(1, 1);
  //Launch the device computation threads
  MatrixMulKernel<<<dimGrid,dimBlock>>>(Md, Nd, Pd, width);
  //Transfer P from device to host
  hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
  //Free device matrices
  hipFree(Md);
  hipFree(Nd);
  hipFree(Pd);
}
int main(){
  const int width = 5;
  float M[width*width], N[width*width], P[width*width];
  srand(time(0));
  for(int i =0; i< (width*width); ++i){
    M[i] = 1+rand()%(101-1);
    N[i] = 1+rand()%(101-1);
    P[i] = 0;
  }
  MatrixMultiplication(M, N, P, width);
  for(int i = 0; i < (width*width) ; ++i){
    printf("%f \n", P[i]);
  }
  return 0;
}
