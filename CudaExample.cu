
#include <hip/hip_runtime.h>
#include "stdio.h"
#define N 10

//Sum Arrays
__global__ void add(int *x, int *y, int *z){
  int tID = blockIdx.x;
  if (tID < N){
    z[tID] = x[tID] + y[tID];
  }
}
int main(){
  int x[N], y[N], z[N];
  int *dev_x, *dev_y, *dev_z;
  /*Allocates size bytes of linear memory on the device and returns in *devPtr a pointer to the allocated memory.
    -> cudaMalloc ( void** devPtr, size_t size) 
  */
  hipMalloc((void **) &dev_x, N*sizeof(int));
  hipMalloc((void **) &dev_y, N*sizeof(int));
  hipMalloc((void **) &dev_z, N*sizeof(int));
  //Fill Arrays
  for (int i = 0; i < N; i++){
    x[i] = i;
    y[i] = i+i;
  }
  /* Copies count bytes from the memory area pointed to by src to the memory area pointed to by dst,specifies the
     direction of the copy. The memory areas may not overlap.
     -> cudaMemcpy(void* dst, const void* src, size_t count, enum cudaMemcpyKind kind) 
  */
  hipMemcpy(dev_x, x, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_y, y, N*sizeof(int), hipMemcpyHostToDevice);
  add<<<N,1>>>(dev_x, dev_y, dev_z);
  hipMemcpy(z, dev_z, N*sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++){
    printf("%d + %d = %d\n", x[i], y[i], z[i]);
  }
  
  return 0;
}
